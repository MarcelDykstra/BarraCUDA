#include <stdio.h>
#include "barracuda.h"

//==============================================================================
void cuLatticeClear(void)
{
  unsigned int n;

  hipFree(state.dev_map);

  hipFree(state.dev_vx);
  hipFree(state.dev_vy);
  hipFree(state.dev_rho);

  for (n = 0; n < N_FLUID; n++) {
    hipFree(state.dev_dist1[n].fC);
    hipFree(state.dev_dist1[n].fE);
    hipFree(state.dev_dist1[n].fW);
    hipFree(state.dev_dist1[n].fS);
    hipFree(state.dev_dist1[n].fN);
    hipFree(state.dev_dist1[n].fNE);
    hipFree(state.dev_dist1[n].fNW);
    hipFree(state.dev_dist1[n].fSE);
    hipFree(state.dev_dist1[n].fSW);

    hipFree(state.dev_dist2[n].fC);
    hipFree(state.dev_dist2[n].fE);
    hipFree(state.dev_dist2[n].fW);
    hipFree(state.dev_dist2[n].fS);
    hipFree(state.dev_dist2[n].fN);
    hipFree(state.dev_dist2[n].fNE);
    hipFree(state.dev_dist2[n].fNW);
    hipFree(state.dev_dist2[n].fSE);
    hipFree(state.dev_dist2[n].fSW);
  }

}

//==============================================================================
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, const mxArray *prhs[])
{
  // Check for proper number of arguments
  if (nrhs != 1) {
    mexErrMsgTxt("One input required.");
  } else if (nlhs > 0) {
    mexErrMsgTxt("Too many output arguments.");
  }

  if (!readState(prhs)) mexErrMsgTxt("Invalid CUDA handle.");

  cuLatticeClear();

  destroyState();
}

