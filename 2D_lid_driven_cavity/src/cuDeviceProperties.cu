#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

//------------------------------------------------------------------------------
extern "C" int cuDeviceCount(void)
{
    int count;
    hipGetDeviceCount(&count);
    return count;
}

//------------------------------------------------------------------------------
extern "C" hipDeviceProp_t cuDeviceProperties(void)
{
    int count;
    hipDeviceProp_t prop;
    hipGetDeviceCount(&count);
    for (int i=0; i < count; i++) {
        hipGetDeviceProperties(&prop, i);
    }
    return prop;
}

