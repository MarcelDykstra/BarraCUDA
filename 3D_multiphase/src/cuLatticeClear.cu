#include <stdio.h>
#include "barracuda.h"

//------------------------------------------------------------------------------
void cuLatticeClear(void)
{
    unsigned int n;

    hipFree(state.dev_map);

    hipFree(state.dev_vx);
    hipFree(state.dev_vy);
    hipFree(state.dev_vz);
    hipFree(state.dev_rho);

    for (n = 0; n < N_FLUID; n++) {
        hipFree(state.dev_dist1[n].fC);
        hipFree(state.dev_dist1[n].fE);
        hipFree(state.dev_dist1[n].fW);
        hipFree(state.dev_dist1[n].fS);
        hipFree(state.dev_dist1[n].fN);
        hipFree(state.dev_dist1[n].fNE);
        hipFree(state.dev_dist1[n].fNW);
        hipFree(state.dev_dist1[n].fSE);
        hipFree(state.dev_dist1[n].fSW);
        hipFree(state.dev_dist1[n].fU);
        hipFree(state.dev_dist1[n].fUE);
        hipFree(state.dev_dist1[n].fUW);
        hipFree(state.dev_dist1[n].fUN);
        hipFree(state.dev_dist1[n].fUS);
        hipFree(state.dev_dist1[n].fD);
        hipFree(state.dev_dist1[n].fDE);
        hipFree(state.dev_dist1[n].fDW);
        hipFree(state.dev_dist1[n].fDN);
        hipFree(state.dev_dist1[n].fDS);

        hipFree(state.dev_dist2[n].fC);
        hipFree(state.dev_dist2[n].fE);
        hipFree(state.dev_dist2[n].fW);
        hipFree(state.dev_dist2[n].fS);
        hipFree(state.dev_dist2[n].fN);
        hipFree(state.dev_dist2[n].fNE);
        hipFree(state.dev_dist2[n].fNW);
        hipFree(state.dev_dist2[n].fSE);
        hipFree(state.dev_dist2[n].fSW);
        hipFree(state.dev_dist2[n].fU);
        hipFree(state.dev_dist2[n].fUE);
        hipFree(state.dev_dist2[n].fUW);
        hipFree(state.dev_dist2[n].fUN);
        hipFree(state.dev_dist2[n].fUS);
        hipFree(state.dev_dist2[n].fD);
        hipFree(state.dev_dist2[n].fDE);
        hipFree(state.dev_dist2[n].fDW);
        hipFree(state.dev_dist2[n].fDN);
        hipFree(state.dev_dist2[n].fDS);
    }

}

//------------------------------------------------------------------------------
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, const mxArray *prhs[])
{
    // Check for proper number of arguments
    if (nrhs != 1) {
        mexErrMsgTxt("One input required.");
    } else if (nlhs > 0) {
        mexErrMsgTxt("Too many output arguments.");
    }

    if (!readState(prhs)) mexErrMsgTxt("Invalid CUDA handle.");

    cuLatticeClear();

    destroyState();
}
